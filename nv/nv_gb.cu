#include "hip/hip_runtime.h"
#include "bootrom.h"
#include "gb_defs.h"
#include <stdio.h>
#include <sys/time.h>

#define CHECK_ERR_CUDA(err) if (err != hipSuccess) { printf("%s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
//#define DEBUG

double get_time() {
  struct timeval tv; gettimeofday(&tv, NULL);
  return (tv.tv_sec + tv.tv_usec * 1e-6);
}

// CPU cycles LUT
__device__ u8 mcycles[256] = {
   4, 12,  8,  8,  4,  4,  8,  4, 20,  8,  8,  8,  4,  4,  8,  4,  // 00-0f
   4, 12,  8,  8,  4,  4,  8,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 10-1f
  12, 12,  8,  8,  4,  4,  8,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 20-2f
  12, 12,  8,  8, 12, 12, 12,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 30-3f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 40-4f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 50-5f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 60-6f
   8,  8,  8,  8,  8,  8,  4,  8,  4,  4,  4,  4,  4,  4,  8,  4,  // 70-7f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 80-8f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 90-9f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // a0-af
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // b0-bf
  20, 12, 16, 16, 24, 16,  8, 16, 20, 16, 16,  4, 24, 24,  8, 16,  // c0-cf
  20, 12, 16,  0, 24, 16,  8, 16, 20, 16, 16,  0, 24,  0,  8, 16,  // d0-df
  12, 12,  8,  0,  0, 16,  8, 16, 16,  4, 16,  0,  0,  0,  8, 16,  // e0-ef
  12, 12,  8,  4,  0, 16,  8, 16, 12,  8, 16,  4,  0,  0,  8, 16}; // f0-ff

// 8-bit write
__device__ void w8(gb* g, u16 a, u8 v) {
  switch (a & 0xf000) {
    case 0x8000: // video ram 0x0000-0x0fff
    case 0x9000: // video ram 0x1000-0x1fff
#ifdef DEBUG
      printf("__w8 VRAM %04x %04x %02x\n", a, a & 0x1fff, v);
#endif
      g->vram[a & 0x1fff] = v;
#ifdef DEBUG
      //printf("%04x %04x %02x\n", a, a & 0x1fff, v);
#endif
      break;
    case 0xc000:
    case 0xd000:
    case 0xe000:
    case 0xf000:
#ifdef DEBUG
      //printf("__w8 HRAM %04x %04x %02x\n", a, a & 0x00ff, v);
#endif
      //if (a < 0xfe00) g->ram[a & 0x1fff] = v; else
      //if (a < 0xff00) g->oam[a & 0xff] = v;
      //else {
      if (a >= 0xff00) g->hram[a & 0xff] = v;
      //  if (a == 0xff46) { oam_ram(g); }
      break;
    default:
#ifdef DEBUG
      printf("w8 unhandled %04x\n", a & 0xf000);
#endif
      break;
  }
}
// 16-bit write
__device__ void w16(gb* g, u16 a, u16 v) { w8(g,a,v&0xff); w8(g,a+1,v>>8); }
// 8-bit read
__device__ u8 r8(gb* g, u16 a) {
  if (a < 512) return g->rom[a];
  if (a == 0xff44) return 0x90;
  if (a >= 0xff00 && a <= 0xffff) {
    return g->hram[a & 0xff];
  }
  return 0;
}
// 16-bit read
__device__ u16 r16(gb* g, u16 a) { return ((u16)(r8(g, a+1)) << 8) | (u16)(r8(g, a)); } // read 2 bytes
// operand fetch
__device__ u8 f8(gb *g) { u8 r = r8(g, PC); PC+=1; return r;  } // fetch operand data (byte)
// 16-bit
__device__ u16 f16(gb* g) { u16 r = r16(g, PC); PC+=2; return r; } // fetch operand data (2 bytes)
// 16-bit stack push/pop
__device__ void push16(gb* g, u16 v) { SP -= 2; w16(g, SP, v); } // push onto the stack
__device__ u16 pop16(gb* g) { u16 v = r16(g, SP); SP+=2; return v; } // pop

// opcode handlers
// see: http://www.pastraiser.com/cpu/gameboy/gameboy_opcodes.html for reference
// _TODO: make pretty, i don't like this approach really, but it's easier to debug

// CB extension handler
// OK, this doesn't look nice, it's a hack to get a byte from u16 array with the right endianess
__device__ u8* ptrs(gb* g, u8 idx) {
  u8* _ptrs[8] = {&B, &C, &D, &E, &H, &L, 0, &A}; return _ptrs[idx];
}

#define PTR_REG(x) ptrs(g, (x))
// bitwise ops
// cb
__device__ u8 rlc(gb* g, u8 v) { // rotate left with carry
  u8 c = ((v >> 7) == 0x01); // carry if bit 7 set
  u8 r = (v << 1) | c; // shift and carry previous bit 7 into 0
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 rrc(gb* g, u8 v) { // rotate right with carry
  u8 c = (v & 0x01); // carry if bit 0 set
  u8 r = (v >> 1) | (c << 7); // shift and carry previous bit 0 into 7
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 rl(gb* g, u8 v) { // rotate left
  u8 c = ((v >> 7) == 0x01); // carry if bit 7 set
  u8 r = (0xff & (v << 1)) | fC;      // shift and carry from flags into 0
  if (r==0) {fZ = 1;} else {fZ = 0;}
  fH = 0; fN = 0; fC = c;
  return r;
}

__device__ u8 rr(gb* g, u8 v) { // rotate right
  u8 c = (v & 0x01); // carry if bit 0 set
  u8 r = (v >> 1) | (fC << 7); // shift and carry from flags into 7
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 sla(gb* g, u8 v) { // shift left arithmetic
  u8 c = (v >> 7) & 0x1; // if bit 7 set
  u8 r = (v << 1);
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 sra(gb* g, u8 v) { //shift right arithmetic
  u8 c = (v & 0x1); //if bit 0 set
  u8 r = (v >> 1) | (v & 0x80); // shift and extend sign
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 srl(gb* g, u8 v) { //shift right logical
  u8 c = (v & 0x1); //if bit 0 set
  u8 r = (v >> 1); // shift
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 swap(gb* g, u8 v) {
  fZ = (v==0); fC=0; fN=0; fH=0;
  return ( (v >> 4) | (v << 4) );
}

__device__ void bitchk(gb* g, u8 n, u8 v) {
  u8 r = ((v >> n) & 0x1) == 0;
  //fZ = 0 of bit was 1
  fN = 0; fH = 1; fZ = r;
}

// ops on accumulator (non-CB)
__device__ void rlca(gb* g) { A = rlc(g, A); fZ = 0;}
__device__ void rrca(gb* g) { A = rrc(g, A); fZ = 0;}
__device__ void  rla(gb* g) { A =  rl(g, A); fZ = 0;}
__device__ void  rra(gb* g) { A =  rr(g, A); fZ = 0;}


__device__ u8 inc8(gb* g, u8 v) {
  u8 r = v + 1;
  fZ = (r == 0); fH = ((v & 0x0f) + 1 > 0x0f); fN = 0;
  return r;
}

__device__ u8 dec8(gb* g, u8 v) {
  u8 r = v - 1;
  fZ = (r == 0); fH = ((v & 0x0f) == 0); fN = 1;
  return r;
}

// 8-bit alu ops
__device__ void _add8(gb* g, u8 v, u8 carry) {
  u8 c = carry ? fC : 0;
  u8 r = A;
  r = A + v + c;
  fH = (((A & 0xf) + (v & 0xf) + c) > 0xf) ? 1 : 0;
  fN = 0; fC = (((u16)(A) + (u16)(v) + (u16)(c)) > 0x00ff) ? 1 : 0;
  A = r;
  fZ = (A == 0);
}

__device__ void _sub8(gb * g, u8 v, u8 carry) {
  // use carry?
  u8 c = carry ? fC : 0;
  u8 r = A;
  r = A - v - c;
  // update flags
  fZ = (r == 0); fH = (((A & 0xf) < ((v & 0xf) + c))) ? 1 : 0; fN = 1;
  fC = (((u16)(A) < (u16)(v) + (u16)(c))) ? 1 : 0;
  A = r;
}

__device__ void add8(gb * g, u8 v) { _add8(g, v, 0); }
__device__ void adc8(gb * g, u8 v) { _add8(g, v, 1); }
__device__ void sub8(gb * g, u8 v) { _sub8(g, v, 0); }
__device__ void sbc8(gb * g, u8 v) { _sub8(g, v, 1); }
__device__ void and8(gb * g, u8 v) { A &= v; fZ = (A == 0); fH = 1; fC = 0; fN = 0; }
__device__ void or8 (gb * g, u8 v)  { A |= v; fZ = (A == 0); fH = 0; fC = 0; fN = 0; }
__device__ void xor8(gb * g, u8 v) { A ^= v; fZ = (A == 0); fH = 0; fC = 0; fN = 0; }
__device__ void cp8 (gb * g, u8 v)  { u8 r = A; _sub8(g, v, 0); A = r; }

// 8-bit alu
__device__ void alu(gb *g, u8 op) {
  u8 src_idx = op & 0x7; //last 3 bits are reg#
  u8 src = (((op >> 6) & 0x3) == 3) ? f8(g) : src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  u8 n = (op >> 3) & 0x07;

  switch (n) { // subgroup, bits xxNNNyyy
    case 0: add8(g, src); break; // 00000yyy
    case 1: adc8(g, src); break; // 00001yyy
    case 2: sub8(g, src); break; // 00010yyy
    case 3: sbc8(g, src); break; // 00011yyy
    case 4: and8(g, src); break; // 00100yyy
    case 5: xor8(g, src); break; // 00101yyy
    case 6: or8(g, src);  break; // 00110yyy
    case 7: cp8(g, src);  break; // 00111yyy
  };
}

// 8-bit inc/dec
__device__ void incdec(gb* g, u8 op) {
  u8 n = op & 0x3; //dec/inc
  u8 dst_idx = (op >> 3) & 0x7; //last 3 bits are reg#
  u8 src = dst_idx == 6 ? r8(g, HL) : (*PTR_REG(dst_idx));
  src = n ? dec8(g, src) : inc8(g, src);
  if (dst_idx != 6) *PTR_REG(dst_idx) = src; else w8(g, HL, src);
}

// register - register load
__device__ void ldrr(gb* g, u8 op) {
  u8 src_idx = op & 0x7; //last 3 bits are reg#
  u8 dst_idx = (op >> 3) & 0x7; //last 3 bits are reg#
  u8 src = (((op >> 6) & 0x3) == 0) ? f8(g) : src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  if (dst_idx != 6) *PTR_REG(dst_idx) = src; else w8(g, HL, src);
}

__device__ void cb_ex(gb* g, u8 x) {

#ifdef DEBUG
  //printf("CB_ex %02x\n", x);
#endif

  u8 src_idx = x & 0x7; //last 3 bits are reg#
  u8 src = src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  u8 op_group = (x >> 6) & 0x03;
  u8 n = (x >> 3) & 0x07;
  u8 res = src;

  switch (op_group) {
    case 0:  // opcode == 00xxxyyy
      switch (n) { // subgroup, bits xxNNNyyy
        case 0: res = rlc (g, src);  break; // 00000yyy
        case 1: res = rrc (g, src);  break; // 00001yyy
        case 2: res = rl  (g, src);   break; // 00010yyy
        case 3: res = rr  (g, src);   break; // 00011yyy
        case 4: res = sla (g, src);  break; // 00100yyy
        case 5: res = sra (g, src);  break; // 00101yyy
        case 6: res = swap(g, src); break; // 00110yyy
        case 7: res = srl (g, src);  break; // 00111yyy
      }; break;
    case 1:  // opcode == 01xxxyyy, test bit n
      bitchk(g, n, src); break;
    case 2:  // opcode == 10xxxyyy, clear bit n
      res &= ~(1<<n); break;
    case 3:  // opcode == 11xxxyyy, set bit n
      res |= (1<<n);  break;
  }

  if (src_idx != 6) *PTR_REG(src_idx) = res; else w8(g, HL, res);
  u8 mcycl = (src_idx == 6)  ? 16 : 8; // 16 cycles if hl
  g->cpu_ticks += mcycl;
}

// JUMPS
__device__ void jr  (gb *g) { PC += (s8)(f8(g)); } // jump relative
__device__ void jp  (gb *g) { PC = r16(g, PC); } // jump absolute
__device__ void jphl(gb *g) { PC = HL; }
__device__ void call(gb *g) { push16(g, PC+2); PC=r16(g,PC); } // uncoditional call
__device__ void ret (gb *g) { PC = pop16(g); } // return from call
__device__ void rst (gb *g, u8 v) { push16(g, PC); PC = (u16)(v); } // reset

// 00 - 0f
__device__ void x00(gb *g, u8 op) { /* nop */ }
__device__ void x01(gb *g, u8 op) { BC = r16(g, PC); PC+=2; }
__device__ void x02(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x03(gb *g, u8 op) { BC++; }
__device__ void x04(gb *g, u8 op) { incdec(g, op); }
__device__ void x05(gb *g, u8 op) { incdec(g, op); }
__device__ void x06(gb *g, u8 op) { ldrr(g, op); }
__device__ void x07(gb *g, u8 op) { rlca(g); }
__device__ void x08(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x09(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x0a(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x0b(gb *g, u8 op) { BC--; }
__device__ void x0c(gb *g, u8 op) { incdec(g, op); }
__device__ void x0d(gb *g, u8 op) { incdec(g, op); }
__device__ void x0e(gb *g, u8 op) { ldrr(g, op);                       } // ld c, imm8
__device__ void x0f(gb *g, u8 op) { rrca(g); }
// 10 - 1f
__device__ void x10(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x11(gb *g, u8 op) { DE = r16(g, PC); PC+=2; }
__device__ void x12(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x13(gb *g, u8 op) { DE++; }
__device__ void x14(gb *g, u8 op) { incdec(g, op); }
__device__ void x15(gb *g, u8 op) { incdec(g, op); }
__device__ void x16(gb *g, u8 op) { ldrr(g, op); }
__device__ void x17(gb *g, u8 op) { rla(g); }
__device__ void x18(gb *g, u8 op) { jr(g); } // jr, s8
__device__ void x19(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x1a(gb *g, u8 op) { A = r8(g, DE); }
__device__ void x1b(gb *g, u8 op) { DE--; }
__device__ void x1c(gb *g, u8 op) { incdec(g, op); }
__device__ void x1d(gb *g, u8 op) { incdec(g, op); }
__device__ void x1e(gb *g, u8 op) { ldrr(g, op);                       } // ld e, imm8
__device__ void x1f(gb *g, u8 op) { rra(g); }
// 20 - 2f
__device__ void x20(gb *g, u8 op) { if (!fZ) jr(g); else { PC+=1; }  } // jr nz, s8
__device__ void x21(gb *g, u8 op) { HL = r16(g, PC); PC+=2;          } // ld hl, imm16
__device__ void x22(gb *g, u8 op) { w8(g, HL++, A); }
__device__ void x23(gb *g, u8 op) { HL++; }
__device__ void x24(gb *g, u8 op) { incdec(g, op); }
__device__ void x25(gb *g, u8 op) { incdec(g, op); }
__device__ void x26(gb *g, u8 op) { ldrr(g, op); }
__device__ void x27(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x28(gb *g, u8 op) { if (fZ)  jr(g); else { PC+=1; }   } // jr z, s8
__device__ void x29(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x2a(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x2b(gb *g, u8 op) { HL--; }
__device__ void x2c(gb *g, u8 op) { incdec(g, op); }
__device__ void x2d(gb *g, u8 op) { incdec(g, op); }
__device__ void x2e(gb *g, u8 op) { ldrr(g, op);                       } // ld l, imm8
__device__ void x2f(gb *g, u8 op) { g->unimpl = 1; }
// 30 - 3f
__device__ void x30(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x31(gb *g, u8 op) { SP = r16(g, PC); PC+=2;          } // ld sp, imm16
__device__ void x32(gb *g, u8 op) { w8(g, HL--, A); }
__device__ void x33(gb *g, u8 op) { SP++; }
__device__ void x34(gb *g, u8 op) { incdec(g, op); }
__device__ void x35(gb *g, u8 op) { incdec(g, op); }
__device__ void x36(gb *g, u8 op) { ldrr(g, op); }
__device__ void x37(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x38(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x39(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x3a(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x3b(gb *g, u8 op) { SP--; }
__device__ void x3c(gb *g, u8 op) { incdec(g, op); }
__device__ void x3d(gb *g, u8 op) { incdec(g, op); }
__device__ void x3e(gb *g, u8 op) { ldrr(g, op);                       } // ld a, imm8
__device__ void x3f(gb *g, u8 op) { g->unimpl = 1; }

// these are all reg-reg loads
// 40 - 4f
__device__ void x40(gb *g, u8 op) { ldrr(g, op); }; __device__ void x41(gb *g, u8 op) { ldrr(g, op); };
__device__ void x42(gb *g, u8 op) { ldrr(g, op); }; __device__ void x43(gb *g, u8 op) { ldrr(g, op); };
__device__ void x44(gb *g, u8 op) { ldrr(g, op); }; __device__ void x45(gb *g, u8 op) { ldrr(g, op); };
__device__ void x46(gb *g, u8 op) { ldrr(g, op); }; __device__ void x47(gb *g, u8 op) { ldrr(g, op); };
__device__ void x48(gb *g, u8 op) { ldrr(g, op); }; __device__ void x49(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4f(gb *g, u8 op) { ldrr(g, op); };
// 50 - 5f
__device__ void x50(gb *g, u8 op) { ldrr(g, op); }; __device__ void x51(gb *g, u8 op) { ldrr(g, op); };
__device__ void x52(gb *g, u8 op) { ldrr(g, op); }; __device__ void x53(gb *g, u8 op) { ldrr(g, op); };
__device__ void x54(gb *g, u8 op) { ldrr(g, op); }; __device__ void x55(gb *g, u8 op) { ldrr(g, op); };
__device__ void x56(gb *g, u8 op) { ldrr(g, op); }; __device__ void x57(gb *g, u8 op) { ldrr(g, op); };
__device__ void x58(gb *g, u8 op) { ldrr(g, op); }; __device__ void x59(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5f(gb *g, u8 op) { ldrr(g, op); };
// 60 - 6f
__device__ void x60(gb *g, u8 op) { ldrr(g, op); }; __device__ void x61(gb *g, u8 op) { ldrr(g, op); };
__device__ void x62(gb *g, u8 op) { ldrr(g, op); }; __device__ void x63(gb *g, u8 op) { ldrr(g, op); };
__device__ void x64(gb *g, u8 op) { ldrr(g, op); }; __device__ void x65(gb *g, u8 op) { ldrr(g, op); };
__device__ void x66(gb *g, u8 op) { ldrr(g, op); }; __device__ void x67(gb *g, u8 op) { ldrr(g, op); };
__device__ void x68(gb *g, u8 op) { ldrr(g, op); }; __device__ void x69(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6f(gb *g, u8 op) { ldrr(g, op); };
// 70 - 7f
__device__ void x70(gb *g, u8 op) { ldrr(g, op); }; __device__ void x71(gb *g, u8 op) { ldrr(g, op); };
__device__ void x72(gb *g, u8 op) { ldrr(g, op); }; __device__ void x73(gb *g, u8 op) { ldrr(g, op); };
__device__ void x74(gb *g, u8 op) { ldrr(g, op); }; __device__ void x75(gb *g, u8 op) { ldrr(g, op); };
__device__ void x76(gb *g, u8 op) { /* HALT */   }; __device__ void x77(gb *g, u8 op) { ldrr(g, op); };
__device__ void x78(gb *g, u8 op) { ldrr(g, op); }; __device__ void x79(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7f(gb *g, u8 op) { ldrr(g, op); };

// reg-reg alu
// 80 - 8f
__device__ void x80(gb *g, u8 op) { alu(g, op);  }; __device__ void x81(gb *g, u8 op) { alu(g, op);  };
__device__ void x82(gb *g, u8 op) { alu(g, op);  }; __device__ void x83(gb *g, u8 op) { alu(g, op);  };
__device__ void x84(gb *g, u8 op) { alu(g, op);  }; __device__ void x85(gb *g, u8 op) { alu(g, op);  };
__device__ void x86(gb *g, u8 op) { alu(g, op);  }; __device__ void x87(gb *g, u8 op) { alu(g, op);  };
__device__ void x88(gb *g, u8 op) { alu(g, op);  }; __device__ void x89(gb *g, u8 op) { alu(g, op);  };
__device__ void x8a(gb *g, u8 op) { alu(g, op);  }; __device__ void x8b(gb *g, u8 op) { alu(g, op);  };
__device__ void x8c(gb *g, u8 op) { alu(g, op);  }; __device__ void x8d(gb *g, u8 op) { alu(g, op);  };
__device__ void x8e(gb *g, u8 op) { alu(g, op);  }; __device__ void x8f(gb *g, u8 op) { alu(g, op);  };
// 90 - 9f;
__device__ void x90(gb *g, u8 op) { alu(g, op);  }; __device__ void x91(gb *g, u8 op) { alu(g, op);  };
__device__ void x92(gb *g, u8 op) { alu(g, op);  }; __device__ void x93(gb *g, u8 op) { alu(g, op);  };
__device__ void x94(gb *g, u8 op) { alu(g, op);  }; __device__ void x95(gb *g, u8 op) { alu(g, op);  };
__device__ void x96(gb *g, u8 op) { alu(g, op);  }; __device__ void x97(gb *g, u8 op) { alu(g, op);  };
__device__ void x98(gb *g, u8 op) { alu(g, op);  }; __device__ void x99(gb *g, u8 op) { alu(g, op);  };
__device__ void x9a(gb *g, u8 op) { alu(g, op);  }; __device__ void x9b(gb *g, u8 op) { alu(g, op);  };
__device__ void x9c(gb *g, u8 op) { alu(g, op);  }; __device__ void x9d(gb *g, u8 op) { alu(g, op);  };
__device__ void x9e(gb *g, u8 op) { alu(g, op);  }; __device__ void x9f(gb *g, u8 op) { alu(g, op);  };
// a0 - af;
__device__ void xa0(gb *g, u8 op) { alu(g, op);  }; __device__ void xa1(gb *g, u8 op) { alu(g, op);  };
__device__ void xa2(gb *g, u8 op) { alu(g, op);  }; __device__ void xa3(gb *g, u8 op) { alu(g, op);  };
__device__ void xa4(gb *g, u8 op) { alu(g, op);  }; __device__ void xa5(gb *g, u8 op) { alu(g, op);  };
__device__ void xa6(gb *g, u8 op) { alu(g, op);  }; __device__ void xa7(gb *g, u8 op) { alu(g, op);  };
__device__ void xa8(gb *g, u8 op) { alu(g, op);  }; __device__ void xa9(gb *g, u8 op) { alu(g, op);  };
__device__ void xaa(gb *g, u8 op) { alu(g, op);  }; __device__ void xab(gb *g, u8 op) { alu(g, op);  };
__device__ void xac(gb *g, u8 op) { alu(g, op);  }; __device__ void xad(gb *g, u8 op) { alu(g, op);  };
__device__ void xae(gb *g, u8 op) { alu(g, op);  }; __device__ void xaf(gb *g, u8 op) { alu(g, op);  };
// b0 - bf;
__device__ void xb0(gb *g, u8 op) { alu(g, op);  }; __device__ void xb1(gb *g, u8 op) { alu(g, op);  };
__device__ void xb2(gb *g, u8 op) { alu(g, op);  }; __device__ void xb3(gb *g, u8 op) { alu(g, op);  };
__device__ void xb4(gb *g, u8 op) { alu(g, op);  }; __device__ void xb5(gb *g, u8 op) { alu(g, op);  };
__device__ void xb6(gb *g, u8 op) { alu(g, op);  }; __device__ void xb7(gb *g, u8 op) { alu(g, op);  };
__device__ void xb8(gb *g, u8 op) { alu(g, op);  }; __device__ void xb9(gb *g, u8 op) { alu(g, op);  };
__device__ void xba(gb *g, u8 op) { alu(g, op);  }; __device__ void xbb(gb *g, u8 op) { alu(g, op);  };
__device__ void xbc(gb *g, u8 op) { alu(g, op);  }; __device__ void xbd(gb *g, u8 op) { alu(g, op);  };
__device__ void xbe(gb *g, u8 op) { alu(g, op);  }; __device__ void xbf(gb *g, u8 op) { alu(g, op);  };
// c0 - cf
__device__ void xc0(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc1(gb *g, u8 op) { BC = pop16(g); }
__device__ void xc2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc5(gb *g, u8 op) { push16(g, BC); }
__device__ void xc6(gb *g, u8 op) { alu(g, op); }
__device__ void xc7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc9(gb *g, u8 op) { ret(g); }
__device__ void xca(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xcb(gb *g, u8 op) { cb_ex(g, f8(g)); } // CB extension
__device__ void xcc(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xcd(gb *g, u8 op) { call(g); }
__device__ void xce(gb *g, u8 op) { alu(g, op); }
__device__ void xcf(gb *g, u8 op) { g->unimpl = 1; }
// d0 - df
__device__ void xd0(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd1(gb *g, u8 op) { DE = pop16(g); }
__device__ void xd2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd5(gb *g, u8 op) { push16(g,DE); }
__device__ void xd6(gb *g, u8 op) { alu(g, op); }
__device__ void xd7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xda(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xdb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xdc(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xdd(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xde(gb *g, u8 op) { alu(g, op); }
__device__ void xdf(gb *g, u8 op) { g->unimpl = 1; }
// e0 - ef
__device__ void xe0(gb *g, u8 op) { w8(g, 0xff00 | f8(g), A); }
__device__ void xe1(gb *g, u8 op) { HL = pop16(g); }
__device__ void xe2(gb *g, u8 op) { w8(g, 0xff00 | C, A); }
__device__ void xe3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe5(gb *g, u8 op) { push16(g,HL); }
__device__ void xe6(gb *g, u8 op) { alu(g, op); }
__device__ void xe7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xea(gb *g, u8 op) { w8(g, f16(g), A); }
__device__ void xeb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xec(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xed(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xee(gb *g, u8 op) { alu(g, op); }
__device__ void xef(gb *g, u8 op) { g->unimpl = 1; }
// f0 - ff
__device__ void xf0(gb *g, u8 op) { A = r8(g, 0xff00 | f8(g));    }
__device__ void xf1(gb *g, u8 op) { AF = pop16(g) & 0xfff0; }
__device__ void xf2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf5(gb *g, u8 op) { push16(g,AF); }
__device__ void xf6(gb *g, u8 op) { alu(g, op);; }
__device__ void xf7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfa(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfc(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfd(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfe(gb *g, u8 op) { alu(g, op); }
__device__ void xff(gb *g, u8 op) { g->unimpl = 1; }

// assign ptrs to opcode handlers
// _TODO: make pretty
__device__ void* ops[256] =
{ &x00, &x01, &x02, &x03, &x04, &x05, &x06, &x07, &x08, &x09, &x0a, &x0b, &x0c, &x0d, &x0e, &x0f,
  &x10, &x11, &x12, &x13, &x14, &x15, &x16, &x17, &x18, &x19, &x1a, &x1b, &x1c, &x1d, &x1e, &x1f,
  &x20, &x21, &x22, &x23, &x24, &x25, &x26, &x27, &x28, &x29, &x2a, &x2b, &x2c, &x2d, &x2e, &x2f,
  &x30, &x31, &x32, &x33, &x34, &x35, &x36, &x37, &x38, &x39, &x3a, &x3b, &x3c, &x3d, &x3e, &x3f,
  &x40, &x41, &x42, &x43, &x44, &x45, &x46, &x47, &x48, &x49, &x4a, &x4b, &x4c, &x4d, &x4e, &x4f,
  &x50, &x51, &x52, &x53, &x54, &x55, &x56, &x57, &x58, &x59, &x5a, &x5b, &x5c, &x5d, &x5e, &x5f,
  &x60, &x61, &x62, &x63, &x64, &x65, &x66, &x67, &x68, &x69, &x6a, &x6b, &x6c, &x6d, &x6e, &x6f,
  &x70, &x71, &x72, &x73, &x74, &x75, &x76, &x77, &x78, &x79, &x7a, &x7b, &x7c, &x7d, &x7e, &x7f,
  &x80, &x81, &x82, &x83, &x84, &x85, &x86, &x87, &x88, &x89, &x8a, &x8b, &x8c, &x8d, &x8e, &x8f,
  &x90, &x91, &x92, &x93, &x94, &x95, &x96, &x97, &x98, &x99, &x9a, &x9b, &x9c, &x9d, &x9e, &x9f,
  &xa0, &xa1, &xa2, &xa3, &xa4, &xa5, &xa6, &xa7, &xa8, &xa9, &xaa, &xab, &xac, &xad, &xae, &xaf,
  &xb0, &xb1, &xb2, &xb3, &xb4, &xb5, &xb6, &xb7, &xb8, &xb9, &xba, &xbb, &xbc, &xbd, &xbe, &xbf,
  &xc0, &xc1, &xc2, &xc3, &xc4, &xc5, &xc6, &xc7, &xc8, &xc9, &xca, &xcb, &xcc, &xcd, &xce, &xcf,
  &xd0, &xd1, &xd2, &xd3, &xd4, &xd5, &xd6, &xd7, &xd8, &xd9, &xda, &xdb, &xdc, &xdd, &xde, &xdf,
  &xe0, &xe1, &xe2, &xe3, &xe4, &xe5, &xe6, &xe7, &xe8, &xe9, &xea, &xeb, &xec, &xed, &xee, &xef,
  &xf0, &xf1, &xf2, &xf3, &xf4, &xf5, &xf6, &xf7, &xf8, &xf9, &xfa, &xfb, &xfc, &xfd, &xfe, &xff };

__global__ void exec(gb* _g_regs, u8 *prog, u16* _g_rand, int prog_len, int steps, int num_threads) {
  int i = blockDim.x * blockIdx.x + threadIdx.x; // thread idx

  u8 op = 0;

  if (i < num_threads) {

    gb *g = &_g_regs[i]; // local copy of regs
    g->rom = prog;

    // outer loop
    u16 r = _g_rand[i];


    u16 delay = r;

    for (int j = 0; j < steps; ++j) {

      if (delay > 0) { delay--; continue; }

      #ifdef DEBUG
      //printf("    %5d [%4d, %5d] step %5d op %2x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x\n",i, blockIdx.x, threadIdx.x, j, op, BC, DE, HL, AF, SP, PC);
      #endif

      op = prog[PC]; PC = (PC + 1) % prog_len;
      ((void(*)(gb*,u8))ops[op])(g,op);

      #ifdef DEBUG
      if (g->unimpl == 1 || PC >= 0x00fe) {
        printf(" i %d, b %d, b %d, s %d, !!! unimpl op %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x\n", i, blockIdx.x, threadIdx.x, j, op, BC, DE, HL, AF, SP, PC);
        return;
      }
      #endif
      if (PC == 0x00fe) printf("    %05d [%04d, %05d] step %05d op %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x ff25 %02x\n",i, blockIdx.x, threadIdx.x, j, op, BC, DE, HL, AF, SP, PC, g->hram[0x11]);
      if (j == 45000) printf("    %05d [%04d, %05d] step %05d op %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x ff25 %02x\n",i, blockIdx.x, threadIdx.x, j, op, BC, DE, HL, AF, SP, PC, g->hram[0x11]);
      if (g->unimpl == 1 || PC >= 0x00fe) { return; };
    }
  }
}

int main(int argc, char **argv) {

  hipError_t err = hipSuccess; // for checking CUDA errors

  // Print the vector length to be used, and compute its size
  int num_blocks = 1; int threads_per_block = 1; int prog_len = 512; int steps = 16;

  // override defaults
  if (argc >= 2) num_blocks = atoi(argv[1]);
  if (argc >= 3) threads_per_block = atoi(argv[2]);
  if (argc >= 4) prog_len = atoi(argv[3]);
  if (argc >= 5) steps = atoi(argv[4]);

  int num_threads = num_blocks * threads_per_block;

  printf("  main: running %d blocks * %d threads (%d threads total)\n", num_blocks, threads_per_block, num_threads);

  // allocate gb registers / state
  gb    *h_in_regs   = (gb *) malloc(num_threads * sizeof(gb));
  gb    *h_out_regs  = (gb *) malloc(num_threads * sizeof(gb));
  u16   *h_rand      = (u16*) malloc(num_threads * sizeof(u16));

  // allocate mem for prog
  u8    *h_prog  = (u8 *)   malloc(prog_len * sizeof(u8));

  if (h_rand == NULL || h_in_regs == NULL || h_out_regs == NULL || h_prog == NULL) { fprintf(stderr, "Failed to allocate host mem!\n"); exit(-1); }

  srand(get_time());
  for (int i = 0; i < num_threads; ++i) { h_rand[i] = rand(); }
  memcpy(h_prog, bootrom, 512);

  // alloc gpu mem
  printf("  main: allocating %zu device bytes\n", prog_len * sizeof(u8) + num_threads * sizeof(gb));

  u8 *d_prog = NULL; gb* d_regs = NULL; u16* d_rand = NULL;
  err = hipMalloc((void **)&d_prog, prog_len    * sizeof(u8) ); CHECK_ERR_CUDA(err);
  err = hipMalloc((void **)&d_regs, num_threads * sizeof(gb) ); CHECK_ERR_CUDA(err);
  err = hipMalloc((void **)&d_rand, num_threads * sizeof(u16)); CHECK_ERR_CUDA(err);

  printf("  main: copying host -> device\n");
  err = hipMemcpy(d_prog, h_prog,    sizeof(u8 ) * prog_len,    hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);
  err = hipMemcpy(d_regs, h_in_regs, sizeof(gb ) * num_threads, hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);
  err = hipMemcpy(d_rand, h_rand,    sizeof(u16) * num_threads, hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);

  printf("  main: running kernel\n");
  hipDeviceSynchronize();
  double start_time = get_time();
  exec<<<num_blocks, threads_per_block>>>(d_regs, d_prog, d_rand, prog_len, steps, num_threads);
  hipDeviceSynchronize();

  double walltime = get_time() - start_time;
  err = hipGetLastError(); CHECK_ERR_CUDA(err);
  printf("  main: copying device -> host\n");
  err = hipMemcpy(h_out_regs, d_regs, sizeof(gb) * num_threads, hipMemcpyDeviceToHost); CHECK_ERR_CUDA(err);

  printf("  main: kernel time = %.6f s, %2.6f us/step, %5.3f MHz\n", walltime, 1e6 * (walltime/(steps * num_threads)), ((steps * num_threads)/walltime)/1e6);
  printf("  main: freeing memory\n");

  // free gpu mem
  err = hipFree(d_prog); CHECK_ERR_CUDA(err);
  err = hipFree(d_regs); CHECK_ERR_CUDA(err);
  err = hipFree(d_rand); CHECK_ERR_CUDA(err);

  // free host mem
  free(h_in_regs); free(h_out_regs); free(h_prog); free(h_rand);

  printf("  main: done.\n");

  return 0;
}
